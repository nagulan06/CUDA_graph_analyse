#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define total 317080
#define N 2099732

extern __managed__ int *list;

struct objects
{
    int len;
    int author;
};
extern __managed__ struct objects *pair;

int comparator (const void * a, const void * b)
{
    struct objects *a1 = (struct objects *)a;
    struct objects *a2 = (struct objects *)b;
    if ((*a1).len > (*a2).len)
        return -1;
    else if ((*a1).len < (*a2).len)
        return 1;
    else
        return 0;
}
/*
int comparator (const void* p1, const void* p2)
{ 
     return (*(int*)p2 - *(int*)p1);    
}
*/

__global__ void co_authors(int *list, struct objects *pair)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    {
        for (int i = index; i < N; i += stride)
        {
            int index1 = list[i]-1;
            pair[index1].author = index1+1;
            atomicAdd(&(pair[index1].len), 1);
        }
    }
}

int main()
{
    FILE *in = fopen("authors.txt", "r");
    int x, y;
   hipMallocManaged(&list, 2099732*sizeof(int), (unsigned int)hipMemAttachGlobal);
   hipMemAdvise(list, 2099732*sizeof(int), hipMemAdviseSetAccessedBy, hipCpuDeviceId);

   hipMallocManaged(&pair, 317080*sizeof(struct objects), (unsigned int)hipMemAttachGlobal);
   hipMemAdvise(pair, 317080*sizeof(struct objects), hipMemAdviseSetAccessedBy, hipCpuDeviceId);
  
    char name[52];
    for(int i=0; i<5; i++)
        fgets(name, 52, in);

    int index = 0;
    while(fscanf(in, "%d %d", &x, &y) == 2)
    {
        list[index] = x;
        list[index+1] = y;
        index = index + 2;
    }
    
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

    printf("first = %d, last = %d\n", list[0], list[2099731]);
 
   
    co_authors<<<numBlocks,blockSize>>>(list, pair);
    hipDeviceSynchronize();
    
    qsort(pair, total, sizeof(pair[0]), comparator);
    printf("len1 = %d,  author = %d\n", pair[0].len, pair[0].author);
      
    return 0;
}
