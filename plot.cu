#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#define total 317080
#define N 2099732

extern __managed__ int *list;
extern __managed__ int *plot_data;

// A structure to hold a pair of data (in this case, the author number and the corresponding number of co-authors)
struct objects
{
    int len;
    int author;
};
extern __managed__ struct objects *pair;

// eomparator function to sort the object structure so that we retain the author number as we start based on the number of co-authors
int comparator (const void * a, const void * b)
{
    struct objects *a1 = (struct objects *)a;
    struct objects *a2 = (struct objects *)b;
    if ((*a1).len > (*a2).len)
        return -1;
    else if ((*a1).len < (*a2).len)
        return 1;
    else
        return 0;
}

// this CUDA kernel calculates the number of co-authors for each author in parallel
__global__ void co_authors(int *list, struct objects *pair)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N; i += stride)
      {
          int index1 = list[i]-1;
          pair[index1].author = index1+1;
          atomicAdd(&(pair[index1].len), 1);
      }
}

// Kernel to generate the data to plot (the number of authors with exactly "d" number of co-authors) in parallel
__global__ void plot(struct objects *pair, int *plot_data)
{
     int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < total; i += stride)
      {
        int index1 = pair[i].len;
        atomicAdd(&(plot_data[index1]), 1);
      }
}

int main()
{
    FILE *in = fopen("authors.txt", "r");
    int x, y;
   hipMallocManaged(&list, 2099732*sizeof(int), (unsigned int)hipMemAttachGlobal);
   hipMemAdvise(list, 2099732*sizeof(int), hipMemAdviseSetAccessedBy, hipCpuDeviceId);

   hipMallocManaged(&pair, 317080*sizeof(struct objects), (unsigned int)hipMemAttachGlobal);
   hipMemAdvise(pair, 317080*sizeof(struct objects), hipMemAdviseSetAccessedBy, hipCpuDeviceId);

   // open the file read the contents and store them in an array (list)
    char name[52];
    for(int i=0; i<5; i++)
        fgets(name, 52, in);

    int index = 0;
    while(fscanf(in, "%d %d", &x, &y) == 2)
    {
        list[index] = x;
        list[index+1] = y;
        index = index + 2;
    }
    
    // Number of threads is 256 and number of blocks is calculated based on that
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;

   // Calculate the number of co-authors and it will be stored in the pair
    co_authors<<<numBlocks,blockSize>>>(list, pair);
    hipDeviceSynchronize();
   
    // sort the pair structure
    qsort(pair, total, sizeof(pair[0]), comparator); 

    // print all those authors who have the maximum number of co-authors
    int max = pair[0].len;
    int i = 0;
    while(pair[i].len == max)
    {
        printf("Maximum number of co-authors = %d,  Author = %d\n", pair[i].len, pair[i].author);
        i++;
    }

   hipMallocManaged(&plot_data, max+1*sizeof(int), (unsigned int)hipMemAttachGlobal);
   hipMemAdvise(plot_data, max+1*sizeof(int), hipMemAdviseSetAccessedBy, hipCpuDeviceId);

    // call plot CUDA kernel that generates the data to plot and stores them in the plot_data array
    numBlocks = (total + blockSize - 1) / blockSize;
    plot<<<numBlocks, blockSize>>>(pair, plot_data);
    hipDeviceSynchronize();

    // write the contents in the plot_data array to an output file
    FILE *out = fopen("output.txt", "w");
    for(int i=0; i<max; i++)
    {
        fprintf(out, "%d  %d\n", i, plot_data[i]);
    }
        return 0;
 
}
